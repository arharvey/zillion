#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>

#include <assert.h>

#include "cudaUtils.h"
#include "solver.h"

#define SOLVER_DIAGNOSTICS (0)

namespace Zillion {

// ---------------------------------------------------------------------------
    
__device__
float3&
unpack(float* array, unsigned n)
{
    return *(float3*)&array[n*3];
}


__device__
const float3&
unpack(const float* array, unsigned n)
{
    return *(const float3*)&array[n*3];
}

// ---------------------------------------------------------------------------

inline
int
roundUpToPower2(int v)
{
    // Handle case where v is already a power of 2
    v--;
    
    // Copy highest set bit to all bits below
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    
    // New power of 2
    v++;
    
    return v;
}



__host__
void
calcDims(int& nBlocks, int& nThreads, const int N, const hipDeviceProp_t& prop)
{
    nThreads = std::min(N, prop.maxThreadsPerBlock);
    nBlocks = (N + prop.maxThreadsPerBlock-1) / prop.maxThreadsPerBlock;
}

// ---------------------------------------------------------------------------

__global__
void
accumulateForcesKernel(float3* Fd, unsigned N, float m, float g)
{
    unsigned n = blockIdx.x*blockDim.x + threadIdx.x;
    while(n < N)
    {
        float3& F = Fd[n];

        F.x = 0.0f;
        F.y = m*g;
        F.z = 0.0f;
        
        n += blockDim.x * gridDim.x;
    }
}

__host__
void
accumulateForces(float3* Fd, unsigned N, float m, float g, const hipDeviceProp_t& prop)
{
    int nBlocks, nThreads;
    calcDims(nBlocks, nThreads, N, prop);
    
    accumulateForcesKernel<<<nBlocks, nThreads>>>(Fd, N, m, g);
    cudaCheckLastError();
}


// ---------------------------------------------------------------------------

__global__
void
forwardEulerSolveKernel(float3* Pd, float3* Vd,
                        const float3* P0d, const float3* Fd,
                        unsigned N,
                        float m, float dt)
{
    unsigned n = blockIdx.x*blockDim.x + threadIdx.x;
    while(n < N)
    {
        const float3& F = Fd[n];
        float3& V = Vd[n];
        
        // a = F/m
        
        const float _1_m = 1.0f/m;
        
        V += F * (_1_m * dt);
        
        const float3& P0 = P0d[n];
        float3& P = Pd[n];

        P = P0 + V*dt;
        
        n += blockDim.x * gridDim.x;
    }
}

__host__
void
forwardEulerSolve(float3* Pd, float3* Vd,
                  const float3* prevPd, const float3* Fd,
                  unsigned N, float m, float dt, const hipDeviceProp_t& prop)
{
    int nBlocks, nThreads;
    calcDims(nBlocks, nThreads, N, prop);
    
    forwardEulerSolveKernel<<<nBlocks, nThreads>>>(Pd, Vd, prevPd, Fd, N,
                                                   m, dt);
    cudaCheckLastError();
}


// ---------------------------------------------------------------------------


__global__
void
handlePlaneCollisionsKernel(float3* Pd, float3* Vd, const float3* P0d,
                            unsigned N, float r, float dt, float Cr)
{
    unsigned n = blockIdx.x*blockDim.x + threadIdx.x;
    while(n < N)
    {
        const float3& P0 = P0d[n];
        float3& V = Vd[n];
        float3& P = Pd[n];
        
        const float3 plane = make_float3(0.0f, 1.0f, 0.0f);
        const float d = 0.0f;
        
        const float distanceFromPlane = (plane ^ P) - d - r;
        
        // Have we collided with the plane?
        if(distanceFromPlane <= 1e-6f)
        {
            const float perpSpeed = (V ^ plane);
            
            // Components of velocity perpendicular and tangent to plane
            const float3 Vp = perpSpeed * plane;
            const float3 Vt = V-Vp;
            
            // Bounce or contact?
            V = Vt;
            if(perpSpeed < -0.1f)
                V -= Vp*Cr;
            else
                V *= (1.0f - 0.5f*dt);
            
            P.y = r;
        }
        
        n += blockDim.x * gridDim.x;
    }
    
}


__host__
void
handlePlaneCollisions(float3* Pd, float3* Vd, const float3* P0d,
                      unsigned N, float r, float dt, float Cr,
                      const hipDeviceProp_t& prop)
{
    int nBlocks, nThreads;
    calcDims(nBlocks, nThreads, N, prop);
    
    handlePlaneCollisionsKernel<<<nBlocks, nThreads>>>(Pd, Vd, P0d, N, r, dt, Cr);
    cudaCheckLastError();
}


// ---------------------------------------------------------------------------

__global__
void
populateCollisionGridKernel(int* d_G, int* d_GN, const float3* const d_P,
                        const int N, const float3 origin, 
                        const int3 dims, const float cellSize)
{
    const float M = 1.0f/cellSize;
    
    int n = blockIdx.x*blockDim.x + threadIdx.x;
    while(n < N)
    {
        // Calculate grid index
        float3 I = d_P[n];
        I -= origin;
        I *= M;
        
        const int cellIndex = int(I.x) +
                              int(I.y)*dims.x +
                              int(I.z)*dims.x*dims.y;
        
        const int i = atomicAdd(d_GN+cellIndex, 1);
        if(i < MAX_OCCUPANCY)
        {
            int* out = d_G + cellIndex*MAX_OCCUPANCY;
            out[i] = n;
        }
        
        n += blockDim.x * gridDim.x;
    }
}


__host__
void
populateCollisionGrid(int* d_G, int* d_GN, const float3* const d_P,
                      const int N, const float3 origin, 
                      const int3 dims, const float cellSize,
                      const hipDeviceProp_t& prop)
{
    int nBlocks, nThreads;
    calcDims(nBlocks, nThreads, N, prop);
    
    populateCollisionGridKernel<<<nBlocks, nThreads>>>(d_G, d_GN, d_P, N, 
                                                   origin, dims, cellSize);
    cudaCheckLastError();
}

// ---------------------------------------------------------------------------

__global__
void
fillKernel(float3* Pd, const float3 v)
{
    Pd[threadIdx.x] = v;
}


template<class Op>
__global__
void
float3ReduceKernel(float3* out_d, const float3* in_d)
{
    extern __shared__ float3 sm[];
    
    // Each thread loads one element from the position array into shared mem
    const int tid = threadIdx.x;
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    sm[tid] = in_d[i];

    __syncthreads();
    
    int s = (blockDim.x/2);
 
    for(; s >= 1; s >>= 1)
    {
        if(tid < s)
        {
            float3* a = sm + tid;
            const float3* b = a + s;
            
            Op::doIt(*a, *b);
        }
        
        __syncthreads();
    }
    
    
    // Write result to global memory
    if(tid == 0)
        out_d[blockIdx.x] = sm[0];
};


__host__
void
reduceDims(int& nBlocks, int& nThreads, const int N, const hipDeviceProp_t& prop)
{
    nThreads = std::min(roundUpToPower2(N), prop.maxThreadsPerBlock);
    
    // Kernel assumes that input float3 array has base-2 number of elements
    nThreads = std::max(64, std::min(nThreads, 128));
    
    nBlocks = (N + nThreads-1) / nThreads;
    
    //assert(nBlocks <= prop.maxGridSize[0]);
}


__host__
unsigned
reduceWorkSize(int N, const hipDeviceProp_t& prop)
{
    int total = 0;
    
    for(int n = 0; n < 2; n++)
    {
        int nBlocks, nThreads;
        reduceDims(nBlocks, nThreads, N, prop);
    
        total += nBlocks * nThreads;
        
        N = nBlocks;
    }
    
    return total;
}


template<class Op>
__host__
unsigned
float3ReducePass(float3* d_out, float3* d_in, int N, const hipDeviceProp_t& prop)
{
    int nBlocks = 0, nThreads = 0;
    reduceDims(nBlocks, nThreads, N, prop);
    
    int nResidualThreads = (nBlocks * nThreads) - N;
    
#if SOLVER_DIAGNOSTICS
    std::cout << "N: " << N 
              << ", Blocks: " << nBlocks
              << ", Threads: " << nThreads
              << ", Residual: " << nResidualThreads << std::endl;
#endif
    
    if(nResidualThreads > 0)
    {
        fillKernel<<<1, nResidualThreads>>>(d_in+N, Op::padding);
        cudaCheckLastError();
    }
    
#if SOLVER_DIAGNOSTICS
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
#endif
    
    int nAllocSharedMemPerBlock = nThreads * sizeof(float3);
    float3ReduceKernel<Op><<<nBlocks, nThreads, nAllocSharedMemPerBlock>>>(d_out, d_in);
    cudaCheckLastError();
    
#if SOLVER_DIAGNOSTICS
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float gpuElapsed;
    hipEventElapsedTime(&gpuElapsed, start, stop);
    
    std::cout << "GPU time: " << gpuElapsed << " ms" << std::endl;
#endif
    
    return nBlocks;
};


template<class Op>
__host__
void
float3Reduce(float3& result, float3* d_work, int N, const hipDeviceProp_t& prop)
{
    int nBlocks, nThreads;
    reduceDims(nBlocks, nThreads, N, prop);
    int firstPassSize = nBlocks * nThreads;
    
    float3* d_W[2] = {d_work, d_work + firstPassSize};
    
    int i = 0;
    while(N > 1)
    {
        float3* d_out = d_W[1-i];
        float3* d_in = d_W[i];
        
        N = float3ReducePass<Op>(d_out, d_in, N, prop);
        
        // Alternate buffers
        i = 1-i;
    }
    
    hipMemcpy(&result, d_W[i], sizeof(float3), hipMemcpyDeviceToHost);
}


// ---------------------------------------------------------------------------

struct MinReductionOp
{
    static float3 padding;
    
    static inline
    __device__
    void
    doIt(float3& a, const float3& b)
    {
        {
            const float Xa = a.x, Xb = b.x;
            if(Xb < Xa)
                a.x = Xb;
        }

        {
            const float Ya = a.y, Yb = b.y;
            if(Yb < Ya)
                a.y = Yb;
        }

        {
            const float Za = a.z, Zb = b.z;
            if(Zb < Za)
                a.z = Zb;
        }
    }
};


float3 MinReductionOp::padding = {std::numeric_limits<float>::max(),
                                  std::numeric_limits<float>::max(),
                                  std::numeric_limits<float>::max()};


__host__
void
minFloat3(float3& result, float3* d_work, int N, const hipDeviceProp_t& prop)
{
    float3Reduce<MinReductionOp>(result, d_work, N, prop);
}


// ---------------------------------------------------------------------------

struct MaxReductionOp
{
    static float3 padding;
    
    static inline
    __device__
    void
    doIt(float3& a, const float3& b)
    {
        {
            const float Xa = a.x, Xb = b.x;
            if(Xb > Xa)
                a.x = Xb;
        }

        {
            const float Ya = a.y, Yb = b.y;
            if(Yb > Ya)
                a.y = Yb;
        }

        {
            const float Za = a.z, Zb = b.z;
            if(Zb > Za)
                a.z = Zb;
        }
    }
};


float3 MaxReductionOp::padding = {-std::numeric_limits<float>::max(),
                                  -std::numeric_limits<float>::max(),
                                  -std::numeric_limits<float>::max()};


__host__
void
maxFloat3(float3& result, float3* d_work, int N, const hipDeviceProp_t& prop)
{
    float3Reduce<MaxReductionOp>(result, d_work, N, prop);
}



// ---------------------------------------------------------------------------

struct SumReductionOp
{
    static float3 padding;
    
    static inline
    __device__
    void
    doIt(float3& a, const float3& b)
    {
        a += b;
    }
};


float3 SumReductionOp::padding = {0.0, 0.0, 0.0};


__host__
void
sumFloat3(float3& result, float3* d_work, int N, const hipDeviceProp_t& prop)
{
    float3Reduce<SumReductionOp>(result, d_work, N, prop);
}

// ---------------------------------------------------------------------------

} // END NAMESPACE ZILLION
