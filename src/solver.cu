
#include <hip/hip_runtime.h>
#include <algorithm>

#include <assert.h>

__device__
float3&
unpack(float* array, unsigned n)
{
    return *(float3*)&array[n*3];
}


__device__
const float3&
unpack(const float* array, unsigned n)
{
    return *(const float3*)&array[n*3];
}


inline
__host__
__device__
float3&
operator*=(float3& a, float b)
{
    a.x *= b;
    a.y *= b;
    a.z *= b;
    
    return a;
}


inline
__host__
__device__
float3
operator*(const float3& a, float b)
{
    float3 v;
    v.x = a.x*b;
    v.y = a.y*b;
    v.z = a.z*b;
    
    return v;
}


inline
__host__
__device__
float3
operator*(float a, const float3& b)
{
    float3 v;
    v.x = b.x*a;
    v.y = b.y*a;
    v.z = b.z*a;
    
    return v;
}


inline
__host__
__device__
float3
operator+(const float3& a, const float3& b)
{
    float3 v;
    v.x = a.x+b.x;
    v.y = a.y+b.y;
    v.z = a.z+b.z;
    
    return v;
}


inline
__host__
__device__
float3&
operator+=(float3& a, const float3& b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
    
    return a;
}


inline
__host__
__device__
float3
operator-(const float3& a, const float3& b)
{
    float3 v;
    v.x = a.x-b.x;
    v.y = a.y-b.y;
    v.z = a.z-b.z;
    
    return v;
}


inline
__host__
__device__
float3&
operator-=(float3& a, const float3& b)
{
    a.x -= b.x;
    a.y -= b.y;
    a.z -= b.z;
    
    return a;
}


inline
__host__
__device__
float
operator^(const float3& a, const float3& b)
{
    return a.x*b.x + a.y*b.y + a.z*b.z;
}


inline
__host__
__device__
float
operator^(const float4& a, const float3& b)
{
    return a.x*b.x + a.y*b.y + a.z*b.z + a.w;
}


inline
__host__
__device__
float
operator^(const float3& a, const float4& b)
{
    return a.x*b.x + a.y*b.y + a.z*b.z + b.w;
}


// ---------------------------------------------------------------------------

inline
unsigned
roundUpToPower2(unsigned v)
{
    // Handle case where v is already a power of 2
    v--;
    
    // Copy highest set bit to all bits below
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    
    // New power of 2
    v++;

    return v;
}


// ---------------------------------------------------------------------------

__global__
void
accumulateForcesKernel(float* Fd, unsigned N, float m, float g)
{
    unsigned n = blockIdx.x*blockDim.x + threadIdx.x;
    while(n < N)
    {
        float3& F = unpack(Fd, n);

        F.x = 0.0f;
        F.y = m*g;
        F.z = 0.0f;
        
        n += blockDim.x * gridDim.x;
    }
}

__host__
void
accumulateForces(float* Fd, unsigned N, float m, float g, unsigned nMaxThreadsPerBlock)
{
    dim3 dimBlock( std::min(N, nMaxThreadsPerBlock) );
    dim3 dimGrid( (N + nMaxThreadsPerBlock-1) / nMaxThreadsPerBlock );
    
    accumulateForcesKernel<<<dimGrid, dimBlock>>>(Fd, N, m, g);
}


// ---------------------------------------------------------------------------

__global__
void
forwardEulerSolveKernel(float* Pd, float* Vd,
                        const float* P0d, const float* Fd,
                        unsigned N,
                        float m, float dt)
{
    unsigned n = blockIdx.x*blockDim.x + threadIdx.x;
    while(n < N)
    {
        const float3& F = unpack(Fd, n);
        float3& V = unpack(Vd, n);
        
        // a = F/m
        
        const float _1_m = 1.0f/m;
        
        V += F * (_1_m * dt);
        
        const float3& P0 = unpack(P0d, n);
        float3& P = unpack(Pd, n);

        P = P0 + V*dt;
        
        n += blockDim.x * gridDim.x;
    }
}

__host__
void
forwardEulerSolve(float* Pd, float* Vd,
                  const float* prevPd, const float* Fd,
                  unsigned N, float m, float dt, unsigned nMaxThreadsPerBlock)
{
    dim3 dimBlock( std::min(N, nMaxThreadsPerBlock) );
    dim3 dimGrid( (N + nMaxThreadsPerBlock-1) / nMaxThreadsPerBlock );
    
    forwardEulerSolveKernel<<<dimGrid, dimBlock>>>(Pd, Vd, prevPd, Fd, N,
                                                   m, dt);
}


// ---------------------------------------------------------------------------


__global__
void
handlePlaneCollisionsKernel(float* Pd, float* Vd, const float* P0d,
                            unsigned N, float r, float dt, float Cr)
{
    unsigned n = blockIdx.x*blockDim.x + threadIdx.x;
    while(n < N)
    {
        const float3& P0 = unpack(P0d, n);
        float3& V = unpack(Vd, n);
        float3& P = unpack(Pd, n);
        
        const float3 plane = make_float3(0.0f, 1.0f, 0.0f);
        const float d = 0.0f;
        
        const float distanceFromPlane = (plane ^ P) - d - r;
        
        // Have we collided with the plane?
        if(distanceFromPlane <= 1e-6f)
        {
            const float perpSpeed = (V ^ plane);
            
            // Components of velocity perpendicular and tangent to plane
            const float3 Vp = perpSpeed * plane;
            const float3 Vt = V-Vp;
            
            // Bounce or contact?
            V = Vt;
            if(perpSpeed < -0.1f)
                V -= Vp*Cr;
            else
                V *= (1.0f - 0.5f*dt);
            
            P.y = r;
        }
        
        n += blockDim.x * gridDim.x;
    }
    
}


__host__
void
handlePlaneCollisions(float* Pd, float* Vd, const float* P0d,
                      unsigned N, float r, float dt, float Cr,
                      unsigned nMaxThreadsPerBlock)
{
    dim3 dimBlock( std::min(N, nMaxThreadsPerBlock) );
    dim3 dimGrid( (N + nMaxThreadsPerBlock-1) / nMaxThreadsPerBlock );
    
    handlePlaneCollisionsKernel<<<dimGrid, dimBlock>>>(Pd, Vd, P0d, N, r, dt, Cr);
}

// ---------------------------------------------------------------------------

__global__
void
fillKernel(float* Pd, const float3 v)
{
    float3& dest = unpack(Pd, threadIdx.x);
    
    dest = v;
}


template<class Op>
__global__
void
float3ReduceKernel(float* Pd)
{
    extern __shared__ float sm[];
    
    // Each thread loads one element from the position array into shared mem
    unsigned tid = threadIdx.x * 3;
    unsigned i = (blockIdx.x*blockDim.x + threadIdx.x) * 3;
    
    sm[tid]   = Pd[i];
    sm[tid+1] = Pd[i+1];
    sm[tid+2] = Pd[i+2];

    __syncthreads();
    
    unsigned s = blockDim.x/2;
    s += s<<1;
    
    for(; s >= 3; s >>= 1)
    {
        if(tid < s)
        {
            float* a = sm + tid;
            const float* b = a + s;
            
            Op::doIt(a, b);
        }
        
        __syncthreads();
    }
    
    
    // Write result to global memory
    if(tid == 0)
    {
        unsigned bid = blockIdx.x*3;
        
        Pd[bid]   = sm[0];
        Pd[bid+1] = sm[1];
        Pd[bid+2] = sm[2];
    }
};


__host__
void
reduceDims(unsigned& nBlocks, unsigned& nThreads, 
                 const unsigned N, const hipDeviceProp_t& prop)
{
    unsigned nThreadsPerBlockRaw = std::min(int(N), prop.maxThreadsPerBlock);
    
    nBlocks = (N + prop.maxThreadsPerBlock-1) / prop.maxThreadsPerBlock;
    
    // Kernel assumes that input float3 array has base-2 number of elements
    nThreads = roundUpToPower2(nThreadsPerBlockRaw);
}


__host__
unsigned
reduceWorkSize(const unsigned N, const hipDeviceProp_t& prop)
{
    unsigned nBlocks = 0, nThreads = 0;
    reduceDims(nBlocks, nThreads, N, prop);
    
    return nBlocks * nThreads;
}


template<class Op>
__host__
unsigned
float3ReducePass(float* Pd, unsigned N, const hipDeviceProp_t& prop)
{
    unsigned nBlocks = 0, nThreads = 0;
    reduceDims(nBlocks, nThreads, N, prop);
    
    unsigned nResidualThreads = (nBlocks * nThreads) - N;
    if(nResidualThreads)
        fillKernel<<<dim3(1), dim3(nResidualThreads)>>>(Pd+N*3, Op::padding);
    
    dim3 dimBlock(nThreads);
    dim3 dimGrid(nBlocks);
    
    unsigned nAllocSharedMemPerBlock = nThreads * 3 * sizeof(float);
    assert(nAllocSharedMemPerBlock < prop.sharedMemPerBlock);
    
    float3ReduceKernel<Op><<<dimGrid, dimBlock, nAllocSharedMemPerBlock>>>(Pd);
    
    return nBlocks;
};


template<class Op>
__host__
void
float3Reduce(float* result, float* Pd, unsigned N, const hipDeviceProp_t& prop)
{
    while(N > 1)
        N = float3ReducePass<Op>(Pd, N, prop);
    
    hipMemcpy(result, Pd, 3*sizeof(float), hipMemcpyDeviceToHost);
}


// ---------------------------------------------------------------------------

struct MinReductionOp
{
    static float3 padding;
    
    static inline
    __device__
    void
    doIt(float* a, const float* b)
    {
        {
            const float Xa = a[0], Xb = b[0];
            if(Xb < Xa)
                a[0] = Xb;
        }

        {
            const float Ya = a[1], Yb = b[1];
            if(Yb < Ya)
                a[1] = Yb;
        }

        {
            const float Za = a[2], Zb = b[2];
            if(Zb < Za)
                a[2] = Zb;
        }
    }
};


float3 MinReductionOp::padding = {std::numeric_limits<float>::max(),
                                  std::numeric_limits<float>::max(),
                                  std::numeric_limits<float>::max()};


__host__
void
minFloat3(float* result, float* Pd, unsigned N, const hipDeviceProp_t& prop)
{
    float3Reduce<MinReductionOp>(result, Pd, N, prop);
}


// ---------------------------------------------------------------------------

struct MaxReductionOp
{
    static float3 padding;
    
    static inline
    __device__
    void
    doIt(float* a, const float* b)
    {
        {
            const float Xa = a[0], Xb = b[0];
            if(Xb > Xa)
                a[0] = Xb;
        }

        {
            const float Ya = a[1], Yb = b[1];
            if(Yb > Ya)
                a[1] = Yb;
        }

        {
            const float Za = a[2], Zb = b[2];
            if(Zb > Za)
                a[2] = Zb;
        }
    }
};


float3 MaxReductionOp::padding = {-std::numeric_limits<float>::max(),
                                  -std::numeric_limits<float>::max(),
                                  -std::numeric_limits<float>::max()};


__host__
void
maxFloat3(float* result, float* Pd, unsigned N, const hipDeviceProp_t& prop)
{
    float3Reduce<MaxReductionOp>(result, Pd, N, prop);
}



// ---------------------------------------------------------------------------

struct SumReductionOp
{
    static float3 padding;
    
    static inline
    __device__
    void
    doIt(float* a, const float* b)
    {
        a[0] += b[0];
        a[1] += b[1];
        a[2] += b[2];
    }
};


float3 SumReductionOp::padding = {0.0, 0.0, 0.0};


__host__
void
sumFloat3(float* result, float* Pd, unsigned N, const hipDeviceProp_t& prop)
{
    float3Reduce<SumReductionOp>(result, Pd, N, prop);
}
